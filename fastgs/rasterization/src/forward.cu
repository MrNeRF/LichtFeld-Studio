#include "hip/hip_runtime.h"
#include "buffer_utils.h"
#include "forward.h"
#include "hip/hip_vector_types.h"
#include "kernels_forward.cuh"
#include "rasterization_config.h"
#include "utils.h"
#include <hipcub/hipcub.hpp>
#include <functional>

// sorting is done separately for depth and tile as proposed in https://github.com/m-schuetz/Splatshop
std::tuple<int, int, int, int, int> fast_gs::rasterization::forward(
    std::function<char*(size_t)> per_primitive_buffers_func,
    std::function<char*(size_t)> per_tile_buffers_func,
    std::function<char*(size_t)> per_instance_buffers_func,
    std::function<char*(size_t)> per_bucket_buffers_func,
    const float3* means,
    const float3* scales_raw,
    const float4* rotations_raw,
    const float* opacities_raw,
    const float3* sh_coefficients_0,
    const float3* sh_coefficients_rest,
    const float4* w2c,
    const float3* cam_position,
    float* image,
    float* alpha,
    const int n_primitives,
    const int active_sh_bases,
    const int total_bases_sh_rest,
    const int width,
    const int height,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const float near_, // near and far are macros in windowns
    const float far_) {
    const dim3 grid(div_round_up(width, config::tile_width), div_round_up(height, config::tile_height), 1);
    const dim3 block(config::tile_width, config::tile_height, 1);
    const int n_tiles = grid.x * grid.y;

    char* per_tile_buffers_blob = per_tile_buffers_func(required<PerTileBuffers>(n_tiles));
    PerTileBuffers per_tile_buffers = PerTileBuffers::from_blob(per_tile_buffers_blob, n_tiles);

    static hipStream_t memset_stream = 0;
    if constexpr (!config::debug) {
        static bool memset_stream_initialized = false;
        if (!memset_stream_initialized) {
            hipStreamCreate(&memset_stream);
            memset_stream_initialized = true;
        }
        hipMemsetAsync(per_tile_buffers.instance_ranges, 0, sizeof(uint2) * n_tiles, memset_stream);
    } else
        hipMemset(per_tile_buffers.instance_ranges, 0, sizeof(uint2) * n_tiles);

    char* per_primitive_buffers_blob = per_primitive_buffers_func(required<PerPrimitiveBuffers>(n_primitives));
    PerPrimitiveBuffers per_primitive_buffers = PerPrimitiveBuffers::from_blob(per_primitive_buffers_blob, n_primitives);

    hipMemset(per_primitive_buffers.n_visible_primitives, 0, sizeof(uint));
    hipMemset(per_primitive_buffers.n_instances, 0, sizeof(uint));

    kernels::forward::preprocess_cu<<<div_round_up(n_primitives, config::block_size_preprocess), config::block_size_preprocess>>>(
        means,
        scales_raw,
        rotations_raw,
        opacities_raw,
        sh_coefficients_0,
        sh_coefficients_rest,
        w2c,
        cam_position,
        per_primitive_buffers.depth_keys.Current(),
        per_primitive_buffers.primitive_indices.Current(),
        per_primitive_buffers.n_touched_tiles,
        per_primitive_buffers.screen_bounds,
        per_primitive_buffers.mean2d,
        per_primitive_buffers.conic_opacity,
        per_primitive_buffers.color,
        per_primitive_buffers.n_visible_primitives,
        per_primitive_buffers.n_instances,
        n_primitives,
        grid.x,
        grid.y,
        active_sh_bases,
        total_bases_sh_rest,
        static_cast<float>(width),
        static_cast<float>(height),
        fx,
        fy,
        cx,
        cy,
        near_,
        far_);
    CHECK_CUDA(config::debug, "preprocess")

    int n_visible_primitives;
    hipMemcpy(&n_visible_primitives, per_primitive_buffers.n_visible_primitives, sizeof(uint), hipMemcpyDeviceToHost);
    int n_instances;
    hipMemcpy(&n_instances, per_primitive_buffers.n_instances, sizeof(uint), hipMemcpyDeviceToHost);

    hipcub::DeviceRadixSort::SortPairs(
        per_primitive_buffers.cub_workspace,
        per_primitive_buffers.cub_workspace_size,
        per_primitive_buffers.depth_keys,
        per_primitive_buffers.primitive_indices,
        n_visible_primitives);
    CHECK_CUDA(config::debug, "hipcub::DeviceRadixSort::SortPairs (Depth)")

    kernels::forward::apply_depth_ordering_cu<<<div_round_up(n_visible_primitives, config::block_size_apply_depth_ordering), config::block_size_apply_depth_ordering>>>(
        per_primitive_buffers.primitive_indices.Current(),
        per_primitive_buffers.n_touched_tiles,
        per_primitive_buffers.offset,
        n_visible_primitives);
    CHECK_CUDA(config::debug, "apply_depth_ordering")

    hipcub::DeviceScan::ExclusiveSum(
        per_primitive_buffers.cub_workspace,
        per_primitive_buffers.cub_workspace_size,
        per_primitive_buffers.offset,
        per_primitive_buffers.offset,
        n_visible_primitives);
    CHECK_CUDA(config::debug, "hipcub::DeviceScan::ExclusiveSum (Primitive Offsets)")

    char* per_instance_buffers_blob = per_instance_buffers_func(required<PerInstanceBuffers>(n_instances));
    PerInstanceBuffers per_instance_buffers = PerInstanceBuffers::from_blob(per_instance_buffers_blob, n_instances);

    kernels::forward::create_instances_cu<<<div_round_up(n_visible_primitives, config::block_size_create_instances), config::block_size_create_instances>>>(
        per_primitive_buffers.primitive_indices.Current(),
        per_primitive_buffers.offset,
        per_primitive_buffers.screen_bounds,
        per_primitive_buffers.mean2d,
        per_primitive_buffers.conic_opacity,
        per_instance_buffers.keys.Current(),
        per_instance_buffers.primitive_indices.Current(),
        grid.x,
        n_visible_primitives);
    CHECK_CUDA(config::debug, "create_instances")

    hipcub::DeviceRadixSort::SortPairs(
        per_instance_buffers.cub_workspace,
        per_instance_buffers.cub_workspace_size,
        per_instance_buffers.keys,
        per_instance_buffers.primitive_indices,
        n_instances);
    CHECK_CUDA(config::debug, "hipcub::DeviceRadixSort::SortPairs (Tile)")

    if constexpr (!config::debug)
        hipStreamSynchronize(memset_stream);

    if (n_instances > 0) {
        kernels::forward::extract_instance_ranges_cu<<<div_round_up(n_instances, config::block_size_extract_instance_ranges), config::block_size_extract_instance_ranges>>>(
            per_instance_buffers.keys.Current(),
            per_tile_buffers.instance_ranges,
            n_instances);
        CHECK_CUDA(config::debug, "extract_instance_ranges")
    }

    kernels::forward::extract_bucket_counts<<<div_round_up(n_tiles, config::block_size_extract_bucket_counts), config::block_size_extract_bucket_counts>>>(
        per_tile_buffers.instance_ranges,
        per_tile_buffers.n_buckets,
        n_tiles);
    CHECK_CUDA(config::debug, "extract_bucket_counts")

    hipcub::DeviceScan::InclusiveSum(
        per_tile_buffers.cub_workspace,
        per_tile_buffers.cub_workspace_size,
        per_tile_buffers.n_buckets,
        per_tile_buffers.bucket_offsets,
        n_tiles);
    CHECK_CUDA(config::debug, "hipcub::DeviceScan::InclusiveSum (Bucket Counts)")

    int n_buckets;
    hipMemcpy(&n_buckets, per_tile_buffers.bucket_offsets + n_tiles - 1, sizeof(uint), hipMemcpyDeviceToHost);

    char* per_bucket_buffers_blob = per_bucket_buffers_func(required<PerBucketBuffers>(n_buckets));
    PerBucketBuffers per_bucket_buffers = PerBucketBuffers::from_blob(per_bucket_buffers_blob, n_buckets);

    kernels::forward::blend_cu<<<grid, block>>>(
        per_tile_buffers.instance_ranges,
        per_tile_buffers.bucket_offsets,
        per_instance_buffers.primitive_indices.Current(),
        per_primitive_buffers.mean2d,
        per_primitive_buffers.conic_opacity,
        per_primitive_buffers.color,
        image,
        alpha,
        per_tile_buffers.max_n_contributions,
        per_tile_buffers.n_contributions,
        per_bucket_buffers.tile_index,
        per_bucket_buffers.color_transmittance,
        width,
        height,
        grid.x);
    CHECK_CUDA(config::debug, "blend")

    return {n_visible_primitives, n_instances, n_buckets, per_primitive_buffers.primitive_indices.selector, per_instance_buffers.primitive_indices.selector};
}
