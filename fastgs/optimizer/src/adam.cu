#include "adam.h"
#include "adam_kernels.cuh"
#include "optimizer_config.h"
#include "utils.h"

void fast_gs::optimizer::adam_step(
    float* param,
    float* exp_avg,
    float* exp_avg_sq,
    const float* param_grad,
    const int n_elements,
    const float lr,
    const float beta1,
    const float beta2,
    const float eps,
    const float bias_correction1_rcp,
    const float bias_correction2_sqrt_rcp)
{
    kernels::adam::adam_step_cu<<<div_round_up(n_elements, config::block_size_adam_step), config::block_size_adam_step>>>(
        param,
        exp_avg,
        exp_avg_sq,
        param_grad,
        n_elements,
        lr,
        beta1,
        beta2,
        eps,
        bias_correction1_rcp,
        bias_correction2_sqrt_rcp
    );
    CHECK_CUDA(config::debug, "adam step")

}
