#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2025 LichtFeld Studio Authors
 *
 * SPDX-License-Identifier: GPL-3.0-or-later */

#include "buffer_utils.h"
#include "forward.h"
#include "hip/hip_vector_types.h"
#include "kernels_forward.cuh"
#include "rasterization_config.h"
#include "utils.h"
#include <hipcub/hipcub.hpp>
#include <functional>

// sorting is done separately for depth and tile as proposed in https://github.com/m-schuetz/Splatshop
void gs::rendering::forward(
    std::function<char*(size_t)> per_primitive_buffers_func,
    std::function<char*(size_t)> per_tile_buffers_func,
    std::function<char*(size_t)> per_instance_buffers_func,
    const float3* means,
    const float3* scales_raw,
    const float4* rotations_raw,
    const float* opacities_raw,
    const float3* sh_coefficients_0,
    const float3* sh_coefficients_rest,
    const float4* w2c,
    const float3* cam_position,
    float* image,
    float* alpha,
    const int n_primitives,
    const int active_sh_bases,
    const int total_bases_sh_rest,
    const int width,
    const int height,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const float near_, // near and far are macros in windowns
    const float far_) {
    const dim3 grid(div_round_up(width, config::tile_width), div_round_up(height, config::tile_height), 1);
    const dim3 block(config::tile_width, config::tile_height, 1);
    const int n_tiles = grid.x * grid.y;

    char* per_tile_buffers_blob = per_tile_buffers_func(required<PerTileBuffers>(n_tiles));
    PerTileBuffers per_tile_buffers = PerTileBuffers::from_blob(per_tile_buffers_blob, n_tiles);

    static hipStream_t memset_stream = 0;
    if constexpr (!config::debug) {
        static bool memset_stream_initialized = false;
        if (!memset_stream_initialized) {
            hipStreamCreate(&memset_stream);
            memset_stream_initialized = true;
        }
        hipMemsetAsync(per_tile_buffers.instance_ranges, 0, sizeof(uint2) * n_tiles, memset_stream);
    } else
        hipMemset(per_tile_buffers.instance_ranges, 0, sizeof(uint2) * n_tiles);

    char* per_primitive_buffers_blob = per_primitive_buffers_func(required<PerPrimitiveBuffers>(n_primitives));
    PerPrimitiveBuffers per_primitive_buffers = PerPrimitiveBuffers::from_blob(per_primitive_buffers_blob, n_primitives);

    hipMemset(per_primitive_buffers.n_visible_primitives, 0, sizeof(uint));
    hipMemset(per_primitive_buffers.n_instances, 0, sizeof(uint));

    kernels::forward::preprocess_cu<<<div_round_up(n_primitives, config::block_size_preprocess), config::block_size_preprocess>>>(
        means,
        scales_raw,
        rotations_raw,
        opacities_raw,
        sh_coefficients_0,
        sh_coefficients_rest,
        w2c,
        cam_position,
        per_primitive_buffers.depth_keys.Current(),
        per_primitive_buffers.primitive_indices.Current(),
        per_primitive_buffers.n_touched_tiles,
        per_primitive_buffers.screen_bounds,
        per_primitive_buffers.mean2d,
        per_primitive_buffers.conic_opacity,
        per_primitive_buffers.color,
        per_primitive_buffers.n_visible_primitives,
        per_primitive_buffers.n_instances,
        n_primitives,
        grid.x,
        grid.y,
        active_sh_bases,
        total_bases_sh_rest,
        static_cast<float>(width),
        static_cast<float>(height),
        fx,
        fy,
        cx,
        cy,
        near_,
        far_);
    CHECK_CUDA(config::debug, "preprocess")

    int n_visible_primitives;
    hipMemcpy(&n_visible_primitives, per_primitive_buffers.n_visible_primitives, sizeof(uint), hipMemcpyDeviceToHost);
    int n_instances;
    hipMemcpy(&n_instances, per_primitive_buffers.n_instances, sizeof(uint), hipMemcpyDeviceToHost);

    hipcub::DeviceRadixSort::SortPairs(
        per_primitive_buffers.cub_workspace,
        per_primitive_buffers.cub_workspace_size,
        per_primitive_buffers.depth_keys,
        per_primitive_buffers.primitive_indices,
        n_visible_primitives);
    CHECK_CUDA(config::debug, "hipcub::DeviceRadixSort::SortPairs (Depth)")

    kernels::forward::apply_depth_ordering_cu<<<div_round_up(n_visible_primitives, config::block_size_apply_depth_ordering), config::block_size_apply_depth_ordering>>>(
        per_primitive_buffers.primitive_indices.Current(),
        per_primitive_buffers.n_touched_tiles,
        per_primitive_buffers.offset,
        n_visible_primitives);
    CHECK_CUDA(config::debug, "apply_depth_ordering")

    hipcub::DeviceScan::ExclusiveSum(
        per_primitive_buffers.cub_workspace,
        per_primitive_buffers.cub_workspace_size,
        per_primitive_buffers.offset,
        per_primitive_buffers.offset,
        n_visible_primitives);
    CHECK_CUDA(config::debug, "hipcub::DeviceScan::ExclusiveSum (Primitive Offsets)")

    char* per_instance_buffers_blob = per_instance_buffers_func(required<PerInstanceBuffers>(n_instances));
    PerInstanceBuffers per_instance_buffers = PerInstanceBuffers::from_blob(per_instance_buffers_blob, n_instances);

    kernels::forward::create_instances_cu<<<div_round_up(n_visible_primitives, config::block_size_create_instances), config::block_size_create_instances>>>(
        per_primitive_buffers.primitive_indices.Current(),
        per_primitive_buffers.offset,
        per_primitive_buffers.screen_bounds,
        per_primitive_buffers.mean2d,
        per_primitive_buffers.conic_opacity,
        per_instance_buffers.keys.Current(),
        per_instance_buffers.primitive_indices.Current(),
        grid.x,
        n_visible_primitives);
    CHECK_CUDA(config::debug, "create_instances")

    hipcub::DeviceRadixSort::SortPairs(
        per_instance_buffers.cub_workspace,
        per_instance_buffers.cub_workspace_size,
        per_instance_buffers.keys,
        per_instance_buffers.primitive_indices,
        n_instances);
    CHECK_CUDA(config::debug, "hipcub::DeviceRadixSort::SortPairs (Tile)")

    if constexpr (!config::debug)
        hipStreamSynchronize(memset_stream);

    if (n_instances > 0) {
        kernels::forward::extract_instance_ranges_cu<<<div_round_up(n_instances, config::block_size_extract_instance_ranges), config::block_size_extract_instance_ranges>>>(
            per_instance_buffers.keys.Current(),
            per_tile_buffers.instance_ranges,
            n_instances);
        CHECK_CUDA(config::debug, "extract_instance_ranges")
    }

    kernels::forward::blend_cu<<<grid, block>>>(
        per_tile_buffers.instance_ranges,
        per_instance_buffers.primitive_indices.Current(),
        per_primitive_buffers.mean2d,
        per_primitive_buffers.conic_opacity,
        per_primitive_buffers.color,
        image,
        alpha,
        width,
        height,
        grid.x);
    CHECK_CUDA(config::debug, "blend")
}