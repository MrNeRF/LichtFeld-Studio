#include "hip/hip_runtime.h"
#include "debug_utils.cuh"
#include "gaussian.cuh"
#include "loss_utils.cuh"
#include "parameters.cuh"
#include "render_utils.cuh"
#include "scene.cuh"
#include <args.hxx>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <random>
#include <torch/torch.h>

void Write_model_parameters_to_file(const ModelParameters& params) {
    std::filesystem::path outputPath = params.output_path;
    std::filesystem::create_directories(outputPath); // Make sure the directory exists

    std::ofstream cfg_log_f(outputPath / "cfg_args");
    if (!cfg_log_f.is_open()) {
        std::cerr << "Failed to open file for writing!" << std::endl;
        return;
    }

    // Write the parameters in the desired format
    cfg_log_f << "Namespace(";
    cfg_log_f << "eval=" << (params.eval ? "True" : "False") << ", ";
    cfg_log_f << "images='" << params.images << "', ";
    cfg_log_f << "model_path='" << params.output_path.string() << "', ";
    cfg_log_f << "resolution=" << params.resolution << ", ";
    cfg_log_f << "sh_degree=" << params.sh_degree << ", ";
    cfg_log_f << "source_path='" << params.source_path.string() << "', ";
    cfg_log_f << "white_background=" << (params.white_background ? "True" : "False") << ")";
    cfg_log_f.close();

    std::cout << "Output folder: " << params.output_path.string() << std::endl;
}

std::vector<int> get_random_indices(int max_index) {
    std::vector<int> indices(max_index);
    std::iota(indices.begin(), indices.end(), 0);
    // Shuffle the vector
    std::shuffle(indices.begin(), indices.end(), std::default_random_engine());
    return indices;
}

int parse_cmd_line_args(const std::vector<std::string>& args,
                        ModelParameters& modelParams,
                        OptimizationParameters& optimParams,
                        PipelineParameters& pipelineParams) {
    if (args.empty()) {
        std::cerr << "No command line arguments provided!" << std::endl;
        return -1;
    }
    args::ArgumentParser parser("3D Gaussian Splatting CUDA Implementation\n",
                                "This program provides a lightning-fast CUDA implementation of the 3D Gaussian Splatting algorithm for real-time radiance field rendering.");
    args::HelpFlag help(parser, "help", "Display this help menu", {'h', "help"});
    args::ValueFlag<std::string> data_path(parser, "data_path", "Path to the training data", {'d', "data_path"});
    args::ValueFlag<std::string> output_path(parser, "output_path", "Path to the training output", {'o', "output_path"});
    args::ValueFlag<uint32_t> iterations(parser, "iterations", "Number of iterations to train the model", {'i', "iter"});
    args::CompletionFlag completion(parser, {"complete"});

    try {
        parser.Prog(args.front());
        parser.ParseArgs(std::vector<std::string>(args.begin() + 1, args.end()));
    } catch (const args::Completion& e) {
        std::cout << e.what();
        return 0;
    } catch (const args::Help&) {
        std::cout << parser;
        return -1;
    } catch (const args::ParseError& e) {
        std::cerr << e.what() << std::endl;
        std::cerr << parser;
        return -1;
    }

    if (data_path) {
        modelParams.source_path = args::get(data_path);
    } else {
        std::cerr << "No data path provided!" << std::endl;
        return -1;
    }
    std::cout << "ModelParams: " << modelParams.source_path << std::endl;
    if (output_path) {
        modelParams.output_path = args::get(output_path);
    } else {
        std::filesystem::path executablePath = std::filesystem::canonical("/proc/self/exe");
        std::filesystem::path parentDir = executablePath.parent_path().parent_path();
        std::filesystem::path outputDir = parentDir / "output";
        try {
            if (!std::filesystem::create_directory(outputDir)) {
                std::cerr << "Directory already exists! Not overwriting it" << std::endl;
                return -1;
            }
        } catch (...) {
            std::cerr << "Failed to create output directory!" << std::endl;
            return -1;
        }
        modelParams.output_path = outputDir;
    }
    std::cout << "ModelParams: " << modelParams.output_path << std::endl;
    if (iterations) {
        optimParams.iterations = args::get(iterations);
    }
    std::cout << "OptimParams: " << optimParams.iterations << std::endl;
    return 0;
}

int main(int argc, char* argv[]) {
    std::vector<std::string> args;
    args.reserve(argc);

    for (int i = 0; i < argc; ++i) {
        args.emplace_back(argv[i]);
    }
    // TODO: read parameters from JSON file or command line
    auto modelParams = ModelParameters();
    auto optimParams = OptimizationParameters();
    auto pipelineParams = PipelineParameters();
    if (parse_cmd_line_args(args, modelParams, optimParams, pipelineParams) < 0) {
        return -1;
    };
    Write_model_parameters_to_file(modelParams);

    auto gaussians = GaussianModel(modelParams.sh_degree);
    auto scene = Scene(gaussians, modelParams);
    gaussians.Training_setup(optimParams);
    if (!torch::cuda::is_available()) {
        // At the moment, I want to make sure that my GPU is utilized.
        std::cout << "CUDA is not available! Training on CPU." << std::endl;
        exit(-1);
    }
    auto pointType = torch::TensorOptions().dtype(torch::kFloat32);
    auto background = modelParams.white_background ? torch::tensor({1.f, 1.f, 1.f}) : torch::tensor({0.f, 0.f, 0.f}, pointType).to(torch::kCUDA);

    const int camera_count = scene.Get_camera_count();
    std::vector<int> indices;
    int last_status_len = 0;
    auto start_time = std::chrono::steady_clock::now();
    for (int iter = 1; iter < optimParams.iterations + 1; ++iter) {
        if (iter % 1000 == 0) {
            gaussians.One_up_sh_degree();
        }

        if (indices.empty()) {
            indices = get_random_indices(camera_count);
        }
        const int camera_index = indices.back();
        indices.pop_back(); // remove last element to iterate over all cameras randomly
        auto& cam = scene.Get_training_camera(camera_index);
        // Render
        auto [image, viewspace_point_tensor, visibility_filter, radii] = render(cam, gaussians, pipelineParams, background);

        // Loss Computations
        auto gt_image = cam.Get_original_image().to(torch::kCUDA);
        auto l1l = gaussian_splatting::l1_loss(image, gt_image);
        auto loss = (1.f - optimParams.lambda_dssim) * l1l + optimParams.lambda_dssim * (1.f - gaussian_splatting::ssim(image, gt_image));

        // Update status line
        auto cur_time = std::chrono::steady_clock::now();
        std::chrono::duration<double> time_elapsed = cur_time - start_time;
        // XXX shouldn't have to create a new stringstream, but resetting takes multiple calls
        std::stringstream status_line;
        // XXX Use thousand separators, but doesn't work for some reason
        status_line.imbue(std::locale(""));
        status_line
            << "\rIteration: " << std::setw(5) << iter
            << "  Loss: " << std::fixed << std::setw(9) << std::setprecision(6) << loss.item<float>()
            << "  Gaussian splats: " << std::setw(8) << (int)gaussians.Get_xyz().size(0)
            << "  Time: " << std::fixed << std::setw(8) << std::setprecision(3) << time_elapsed.count() << "s"
            << "  Avg iter/s: " << std::fixed << std::setw(4) << std::setprecision(1) << 1.0*iter/time_elapsed.count()
            << "  " // Some extra whitespace, in case a "Pruning ... points" message gets printed after
            ;
        const int curlen = status_line.str().length();
        const int ws = last_status_len - curlen;
        if (ws > 0)
            status_line << std::string(ws, ' ');
        std::cout << status_line.str() << std::flush;
        last_status_len = curlen;

        loss.backward();

        {
            torch::NoGradGuard no_grad;
            auto visible_max_radii = gaussians._max_radii2D.masked_select(visibility_filter);
            auto visible_radii = radii.masked_select(visibility_filter);
            auto max_radii = torch::max(visible_max_radii, visible_radii);
            gaussians._max_radii2D.masked_scatter_(visibility_filter, max_radii);
            if (iter == optimParams.iterations) {
                std::cout << std::endl;
                gaussians.Save_ply(modelParams.output_path, iter, true);
                break;
            }
            if (iter % 7'000 == 0) {
                gaussians.Save_ply(modelParams.output_path, iter, false);
            }

            // that should be the max. Stop iterating.
            if (iter == 30'000) {
                std::cout << std::endl;
                gaussians.Save_ply(modelParams.output_path, iter, true);
                break;
            }

            // Densification
            if (iter < optimParams.densify_until_iter) {
                gaussians.Add_densification_stats(viewspace_point_tensor, visibility_filter);
                if (iter > optimParams.densify_from_iter && iter % optimParams.densification_interval == 0) {
                    // @TODO: Not sure about type
                    float size_threshold = iter > optimParams.opacity_reset_interval ? 20.f : -1.f;
                    gaussians.Densify_and_prune(optimParams.densify_grad_threshold, 0.005f, scene.Get_cameras_extent(), size_threshold);
                }

                if (iter % optimParams.opacity_reset_interval == 0 || (modelParams.white_background && iter == optimParams.densify_from_iter)) {
                    std::cout << "iteration " << iter << " resetting opacity" << std::endl;
                    gaussians.Reset_opacity();
                }
            }

            //  Optimizer step
            if (iter < optimParams.iterations) {
                gaussians._optimizer->step();
                gaussians._optimizer->zero_grad(true);
                // @TODO: Not sure about type
                gaussians.Update_learning_rate(iter);
            }
        }
    }

    auto cur_time = std::chrono::steady_clock::now();
    std::chrono::duration<double> time_elapsed = cur_time - start_time;

    std::cout << std::endl << "All done in "
        << std::fixed << std::setw(7) << std::setprecision(3) << time_elapsed.count() << "s, avg "
        << std::fixed << std::setw(4) << std::setprecision(1) << 1.0*optimParams.iterations/time_elapsed.count() << " iter/s"
        << std::endl;

    return 0;
}
