#include "hip/hip_runtime.h"
#include "config.h"

#ifdef CUDA_GL_INTEROP_ENABLED

#include <cstdint>
#include <hip/hip_runtime.h>

namespace gs {

    // Kernel for converting RGB float to RGBA uint8
    __global__ void convertRGBFloatToRGBAUint8(
        const float* __restrict__ rgb,
        uint8_t* __restrict__ rgba,
        int width, int height) {

        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < width && y < height) {
            int idx = y * width + x;
            int rgb_idx = idx * 3;
            int rgba_idx = idx * 4;

            // Convert float [0,1] to uint8 [0,255] with clamping
            rgba[rgba_idx + 0] = min(255, max(0, __float2int_rn(rgb[rgb_idx + 0] * 255.0f)));
            rgba[rgba_idx + 1] = min(255, max(0, __float2int_rn(rgb[rgb_idx + 1] * 255.0f)));
            rgba[rgba_idx + 2] = min(255, max(0, __float2int_rn(rgb[rgb_idx + 2] * 255.0f)));
            rgba[rgba_idx + 3] = 255; // Alpha
        }
    }

    // Kernel for converting RGBA float to RGBA uint8
    __global__ void convertRGBAFloatToRGBAUint8(
        const float* __restrict__ rgba_in,
        uint8_t* __restrict__ rgba_out,
        int width, int height) {

        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < width && y < height) {
            int idx = (y * width + x) * 4;

            // Convert float [0,1] to uint8 [0,255] with clamping
            rgba_out[idx + 0] = min(255, max(0, __float2int_rn(rgba_in[idx + 0] * 255.0f)));
            rgba_out[idx + 1] = min(255, max(0, __float2int_rn(rgba_in[idx + 1] * 255.0f)));
            rgba_out[idx + 2] = min(255, max(0, __float2int_rn(rgba_in[idx + 2] * 255.0f)));
            rgba_out[idx + 3] = min(255, max(0, __float2int_rn(rgba_in[idx + 3] * 255.0f)));
        }
    }

    // Kernel for flipping image vertically (OpenGL uses bottom-left origin)
    template <typename T>
    __global__ void flipVertical(
        const T* __restrict__ input,
        T* __restrict__ output,
        int width, int height, int channels) {

        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < width && y < height) {
            int flipped_y = height - 1 - y;
            int src_idx = (y * width + x) * channels;
            int dst_idx = (flipped_y * width + x) * channels;

#pragma unroll
            for (int c = 0; c < channels; ++c) {
                output[dst_idx + c] = input[src_idx + c];
            }
        }
    }

} // namespace gs

#endif // CUDA_GL_INTEROP_ENABLED